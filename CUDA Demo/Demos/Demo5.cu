#include <stdio.h>
#include <hip/hip_runtime.h>  
#define N 100
__global__    void fun() {
    printf("%d\n", threadIdx.x*threadIdx.x);
}
int main() {
    fun<<<1, N>>>();
    hipDeviceSynchronize();  
    return 0;
}

