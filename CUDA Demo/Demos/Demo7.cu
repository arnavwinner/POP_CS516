#include <stdio.h>  
#include <hip/hip_runtime.h>  
#define N 100

__global__ void fun(int *a) {
	a[threadIdx.x] = threadIdx.x * threadIdx.x;
}

int main() {
	int a[N], *da;  int i;
	hipMalloc(&da, N * sizeof(int));
	fun<<<1, N>>>(da);
	hipMemcpy(a, da, N * sizeof(int),
	hipMemcpyDeviceToHost);  
	for (i = 0; i < N; ++i)
		printf("%d\n", a[i]);  
	return 0;
}

