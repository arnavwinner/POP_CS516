#include <stdio.h>  
#include <hip/hip_runtime.h>
__global__ void dkernel() {  
    printf("Hello World.\n");
}
int main() {
    dkernel<<<1, 32>>>();  
    hipDeviceSynchronize();  
    return 0;
}

