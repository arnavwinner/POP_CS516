#include <stdio.h>  
#include <hip/hip_runtime.h>
__global__   void dkernel(char *arr, int rrlen) { 	
	unsigned id = threadIdx.x;
	if (id < 5) {
		++arr[id];
	}
}
int main() {
	char cpuarr[] = "CS516",  *gpuarr;
	hipMalloc(&gpuarr, sizeof(char) * (1 + strlen(cpuarr)));
	hipMemcpy(gpuarr, cpuarr, sizeof(char) * (1 + strlen(cpuarr)), hipMemcpyHostToDevice);
	dkernel<<<1, 32>>>(gpuarr, strlen(cpuarr));  hipDeviceSynchronize();	// unnecessary.
	hipMemcpy(cpuarr, gpuarr, sizeof(char) * (1 + strlen(cpuarr)), hipMemcpyDeviceToHost);  
  printf(cpuarr);
	return 0;
}
