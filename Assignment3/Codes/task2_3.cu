#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

const int N = 4194304;
const int threadsPerBlock = 256;
const int arraySize = N;
const int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;

__global__ void reduce0(int *g_idata, int *g_odata) {
  extern __shared__ int sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
  __syncthreads();
  for (unsigned int s=blockDim.x/2; s>0; s>>=1) 
  {
    if (tid < s) {
    sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

float time_diff(struct timeval *start, struct timeval *end) {
  return (end->tv_sec - start->tv_sec) + 1e-6 * (end->tv_usec - start->tv_usec);
}

int main() {
    struct timeval start, end;

    int* h_input = (int*)malloc(arraySize * sizeof(int));
    int* h_output = (int*)malloc(blocksPerGrid * sizeof(int));


    for (int i = 0; i < arraySize; ++i) {
        h_input[i] = i;
    }

    int* d_input, *d_output;
    hipMalloc((void**)&d_input, arraySize * sizeof(int));
    hipMalloc((void**)&d_output, blocksPerGrid * sizeof(int));

    hipMemcpy(d_input, h_input, arraySize * sizeof(int), hipMemcpyHostToDevice);

    
    gettimeofday(&start, NULL);

    reduce0<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output);
    hipDeviceSynchronize();

    gettimeofday(&end, NULL);

    hipMemcpy(h_output, d_output, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

    int result = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        result += h_output[i];
    }

    printf("time spent: %0.8f sec\n", time_diff(&start, &end));

    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}