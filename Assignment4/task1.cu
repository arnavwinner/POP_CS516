#include "hip/hip_runtime.h"
%%cuda

#include <stdio.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16 // Define the tile size for matrix multiplication

__global__ void matrixVectorMulWithoutCoalescing(float *M, float *V, float *Z, int m) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < m) {
        float sum = 0.0f;
        for (int i = 0; i < m; ++i) {
            sum += V[i] * M[i * m + tid];
        }
        Z[tid] = sum;
    }
}

__global__ void matrixVectorMulWithCoalescing(float *M, float *V, float *Z, int m) {
    __shared__ float s_V[TILE_SIZE];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = threadIdx.x;

    if (tid < m) {
        float sum = 0.0f;
        s_V[idx] = V[tid];
        __syncthreads();
        for (int i = 0; i < m; ++i) {
            sum += s_V[i] * M[i * m + tid];
        }
        Z[tid] = sum;
    }
}

void printMatrix(float *matrix, int m) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            printf("%.2f ", matrix[i * m + j]);
        }
        printf("\n");
    }
}

int main() {
    int m = 32; // size of the matrix and vector (reduced for readability)
    int size = m * m * sizeof(float); // size of matrix M
    int vSize = m * sizeof(float); // size of vector V

    // Allocate memory for the host
    float *h_M = (float *)malloc(size);
    float *h_V = (float *)malloc(vSize);
    float *h_Z = (float *)malloc(vSize);

    // Initialize matrix M and vector V
    for (int i = 0; i < m * m; ++i) {
        h_M[i] = 1.0f; // fill matrix M with 1s for simplicity
    }
    for (int i = 0; i < m; ++i) {
        h_V[i] = 1.0f; // fill vector V with 1s for simplicity
    }

    // Allocate memory for the device
    float *d_M, *d_V, *d_Z;
    hipMalloc((void **)&d_M, size);
    hipMalloc((void **)&d_V, vSize);
    hipMalloc((void **)&d_Z, vSize);

    // Copy data from host to device
    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, vSize, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int blockSize = 256;
    int gridSize = (m + blockSize - 1) / blockSize;

    // Perform matrix-vector multiplication without coalesced memory accesses
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    matrixVectorMulWithoutCoalescing<<<gridSize, blockSize>>>(d_M, d_V, d_Z, m);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for matrix-vector multiplication without coalesced memory accesses: %f milliseconds\n", milliseconds);

    // Copy the result back to the host
    hipMemcpy(h_Z, d_Z, vSize, hipMemcpyDeviceToHost);

    // Print the result matrix
    // printf("Result Matrix without coalesced memory accesses:\n");
    // printMatrix(h_Z, m);

    // Free device memory
    hipFree(d_M);
    hipFree(d_V);
    hipFree(d_Z);

    // Perform matrix-vector multiplication with coalesced memory accesses
    hipMalloc((void **)&d_M, size);
    hipMalloc((void **)&d_V, vSize);
    hipMalloc((void **)&d_Z, vSize);

    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, vSize, hipMemcpyHostToDevice);

    hipEventRecord(start);
    
    matrixVectorMulWithCoalescing<<<gridSize, blockSize>>>(d_M, d_V, d_Z, m);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("\nTime for matrix-vector multiplication with coalesced memory accesses: %f milliseconds\n", milliseconds);

    hipMemcpy(h_Z, d_Z, vSize, hipMemcpyDeviceToHost);

    // Print the result matrix
    // printf("Result Matrix with coalesced memory accesses:\n");
    // printMatrix(h_Z, m);

    // Free host memory
    free(h_M);
    free(h_V);
    free(h_Z);

    // Free device memory
    hipFree(d_M);
    hipFree(d_V);
    hipFree(d_Z);

    return 0;
}
